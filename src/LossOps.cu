#include "hip/hip_runtime.h"
#include "LossOps.h"
#include ""
#include <hip/hip_runtime.h>
#include <iostream>

namespace LossOps {

// CUDA Kernels for Loss Computation
__global__ void MSEKernel(const float* yTrue, const float* yPred, float* loss, int size) { 
    extern __shared__ float sharedLoss[];  
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    sharedLoss[threadIdx.x] = 0.0f;

    if (idx < size) {
        float diff = yTrue[idx] - yPred[idx];
        sharedLoss[threadIdx.x] = diff * diff;
    }

    __syncthreads();

    // Parallel reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride /=2) {
        if (threadIdx.x < stride) {
            sharedLoss[threadIdx.x] += sharedLoss[threadIdx.x + stride];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        atomicAdd(loss, sharedLoss[0]);
    }
}

// Kernel for computing error gradients
__global__ void computeErrorKernel(const float* yTrue, const float* yPred, float* error, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        error[idx] = yPred[idx] - yTrue[idx];
    }
}

// Kernel for cross entropy loss computation
__global__ void crossEntropyLossKernel(const float* yTrue, const float* yPred, float* loss, int numClasses, int batchSize) { 
    extern __shared__ float sharedLoss[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    sharedLoss[threadIdx.x] = 0.0f;

    if (idx < batchSize*numClasses) {
        if (yTrue[idx] > 0.0f) {
            sharedLoss[threadIdx.x] = -yTrue[idx] * logf(yPred[idx] + 1e-8);
        }
    }
    __syncthreads();

    // Parallel reduction for loss computation
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride) {
            sharedLoss[threadIdx.x] += sharedLoss[threadIdx.x + stride];
        }
        __syncthreads();
    }
    if(threadIdx.x == 0) {
        atomicAdd(loss, sharedLoss[0]);
    }
}

// Mean Squared Error loss computation
float LossOps::MeanSquaredError(const float* targets, const float* predictions, int size, bool isGPU) {
    float *d_targets, *d_predictions, *d_loss;
     
    if (isGPU) {
        // Use GPU data directly
        hipMalloc(&d_loss, size * sizeof(float));
        
        int blockSize = 256;
        int numBlocks = (size + blockSize - 1) / blockSize;
        MSEKernel<<<numBlocks, blockSize>>>(targets, predictions, d_loss, size);
    } else {
        // Copy data to GPU
        hipMalloc(&d_targets, size * sizeof(float));
        hipMalloc(&d_predictions, size * sizeof(float));
        hipMalloc(&d_loss, size * sizeof(float));

        hipMemcpy(d_targets, targets, size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_predictions, predictions, size * sizeof(float), hipMemcpyHostToDevice);

        int blockSize = 256;
        int numBlocks = (size + blockSize - 1) / blockSize;
        MSEKernel<<<numBlocks, blockSize>>>(d_targets, d_predictions, d_loss, size);
    }

    // Get final loss value
    float totalLoss;
    hipMemcpy(&totalLoss, d_loss, sizeof(float), hipMemcpyDeviceToHost);
    
    // Cleanup GPU memory
    if (!isGPU) {
        hipFree(d_targets);
        hipFree(d_predictions);
    }
    hipFree(d_loss);
    return totalLoss;
}

// Cross Entropy loss computation
float LossOps::gpuCrossEntropyLoss(const float* yTrue, const float* yPred, int batchSize, int numClasses) {
    float* d_yTrue;
    float* d_yPred;
    float* d_loss;
    float h_loss = 0.0f;

    // Allocate and copy data to GPU
    hipMalloc(&d_yTrue, sizeof(float) * batchSize * numClasses);
    hipMalloc(&d_yPred, sizeof(float) * batchSize * numClasses);
    hipMalloc(&d_loss, sizeof(float));

    hipMemcpy(d_yTrue, yTrue, sizeof(float) * batchSize * numClasses, hipMemcpyHostToDevice);
    hipMemcpy(d_yPred, yPred, sizeof(float) * batchSize * numClasses, hipMemcpyHostToDevice);
    hipMemcpy(d_loss, &h_loss, sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (batchSize * numClasses + threadsPerBlock - 1) / threadsPerBlock;
    crossEntropyLossKernel<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(float)>>>(
        d_yTrue, d_yPred, d_loss, numClasses, batchSize);

    // Get result and cleanup
    hipMemcpy(&h_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_yTrue);
    hipFree(d_yPred);
    hipFree(d_loss);

    return h_loss / batchSize;
}

// Compute error gradients for backpropagation
void LossOps::computeError(const float* yTrue, const float* yPred, float* error, int size, bool isGPU) {
    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((size + threadsPerBlock.x - 1) / threadsPerBlock.x);

    if (isGPU) {
        // Use GPU data directly
        computeErrorKernel<<<blocksPerGrid, threadsPerBlock>>>(yTrue, yPred, error, size);
    } else {
        // Copy data to GPU
        float *d_yTrue, *d_yPred;
        hipMalloc(&d_yTrue, size * sizeof(float));
        hipMalloc(&d_yPred, size * sizeof(float));
        
        hipMemcpy(d_yTrue, yTrue, size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_yPred, yPred, size * sizeof(float), hipMemcpyHostToDevice);
        
        computeErrorKernel<<<blocksPerGrid, threadsPerBlock>>>(d_yTrue, d_yPred, error, size);
        
        hipFree(d_yTrue);
        hipFree(d_yPred);
    }
    
    hipDeviceSynchronize();
}
}