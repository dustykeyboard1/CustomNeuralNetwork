#include "hip/hip_runtime.h"
#include "LossOps.h"
#include ""
#include <hip/hip_runtime.h>
#include <iostream>

namespace LossOps {
__global__ void MSEKernel(const float* yTrue, const float* yPred, float* loss, int size) { 
    extern __shared__ float sharedLoss[];  
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    sharedLoss[threadIdx.x] = 0.0f;

    if (idx < size) {
        float diff = yTrue[idx] - yPred[idx];
        sharedLoss[threadIdx.x] = diff * diff;
    }

    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride /=2) {
        if (threadIdx.x < stride) {
            sharedLoss[threadIdx.x] += sharedLoss[threadIdx.x + stride];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        atomicAdd(loss, sharedLoss[0]);
    }
}

__global__ void crossEntropyLossKernel(const float* yTrue, const float* yPred, float* loss, int numClasses, int batchSize) { 
    extern __shared__ float sharedLoss[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    sharedLoss[threadIdx.x] = 0.0f;

    if (idx < batchSize*numClasses) {

        if (yTrue[idx] > 0.0f) {
            sharedLoss[threadIdx.x] = -yTrue[idx] * logf(yPred[idx] + 1e-8);
        }
    }
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride) {
            sharedLoss[threadIdx.x] += sharedLoss[threadIdx.x + stride];
        }
        __syncthreads();
    }
    if(threadIdx.x == 0) {
        atomicAdd(loss, sharedLoss[0]);
    }
}

float LossOps::MeanSquaredError(const float* targets, const float* predictions, int size, bool isGPU) {
    float *d_targets, *d_predictions, *d_loss;
    float totalLoss = 0.0f;

    if (isGPU) {
        // Data is already on GPU, just allocate loss buffer
        hipMalloc(&d_loss, size * sizeof(float));
        
        // Launch kernel directly with GPU data
        int blockSize = 256;
        int numBlocks = (size + blockSize - 1) / blockSize;
        MSEKernel<<<numBlocks, blockSize>>>(targets, predictions, d_loss, size);
    } else {
        // Allocate and copy data to GPU
        hipMalloc(&d_targets, size * sizeof(float));
        hipMalloc(&d_predictions, size * sizeof(float));
        hipMalloc(&d_loss, size * sizeof(float));

        hipMemcpy(d_targets, targets, size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_predictions, predictions, size * sizeof(float), hipMemcpyHostToDevice);

        int blockSize = 256;
        int numBlocks = (size + blockSize - 1) / blockSize;
        MSEKernel<<<numBlocks, blockSize>>>(d_targets, d_predictions, d_loss, size);
    }

    // Reduce on GPU to compute sum
    // ... (reduction kernel to sum all losses)
    
    // Copy final result back
    float meanLoss;
    hipMemcpy(&meanLoss, d_loss, sizeof(float), hipMemcpyDeviceToHost);
    
    // Cleanup
    if (!isGPU) {
        hipFree(d_targets);
        hipFree(d_predictions);
    }
    hipFree(d_loss);

    return meanLoss / size;
}

float LossOps::gpuCrossEntropyLoss(const float* yTrue, const float* yPred, int batchSize, int numClasses) {
    float* d_yTrue;
    float* d_yPred;
    float* d_loss;
    float h_loss = 0.0f;

    hipMalloc(&d_yTrue, sizeof(float) * batchSize * numClasses);
    hipMalloc(&d_yPred, sizeof(float) * batchSize * numClasses);
    hipMalloc(&d_loss, sizeof(float));

    hipMemcpy(d_yTrue, yTrue, sizeof(float) * batchSize * numClasses, hipMemcpyHostToDevice);
    hipMemcpy(d_yPred, yPred, sizeof(float) * batchSize * numClasses, hipMemcpyHostToDevice);
    hipMemcpy(d_loss, &h_loss, sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (batchSize * numClasses + threadsPerBlock - 1) / threadsPerBlock;

    crossEntropyLossKernel<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(float)>>>(d_yTrue, d_yPred, d_loss, numClasses, batchSize);

    hipMemcpy(&h_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_yTrue);
    hipFree(d_yPred);
    hipFree(d_loss);

    return h_loss / batchSize;
}


}