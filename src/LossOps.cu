#include "hip/hip_runtime.h"
#include "LossOps.h"
#include ""
#include <hip/hip_runtime.h>
#include <iostream>

namespace LossOps {
__global__ void MSEKernel(const float* yTrue, const float* yPred, float* loss, int size) { 
    extern __shared__ float sharedLoss[];  
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    sharedLoss[threadIdx.x] = 0.0f;

    if (idx < size) {
        float diff = yTrue[idx] - yPred[idx];
        sharedLoss[threadIdx.x] = diff * diff;
    }

    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride /=2) {
        if (threadIdx.x < stride) {
            sharedLoss[threadIdx.x] += sharedLoss[threadIdx.x + stride];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        atomicAdd(loss, sharedLoss[0]);
    }
}

__global__ void crossEntropyLossKernel(const float* yTrue, const float* yPred, float* loss, int numClasses, int batchSize) { 
    extern __shared__ float sharedLoss[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    sharedLoss[threadIdx.x] = 0.0f;

    if (idx < batchSize*numClasses) {

        if (yTrue[idx] > 0.0f) {
            sharedLoss[threadIdx.x] = -yTrue[idx] * logf(yPred[idx] + 1e-8);
        }
    }
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride) {
            sharedLoss[threadIdx.x] += sharedLoss[threadIdx.x + stride];
        }
        __syncthreads();
    }
    if(threadIdx.x == 0) {
        atomicAdd(loss, sharedLoss[0]);
    }
}

float MeanSquaredError(const float* yTrue, const float* yPred, int size) {
    float* d_yTrue; 
    float* d_yPred;
    float* d_loss;
    float h_loss = 0.0f;

    hipMalloc(&d_yTrue, sizeof(float) * size);
    hipMalloc(&d_yPred, sizeof(float) * size);
    hipMalloc(&d_loss, sizeof(float));

    hipMemcpy(d_yTrue, yTrue, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_yPred, yPred, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(d_loss, &h_loss, sizeof(float), hipMemcpyHostToDevice);

    // Configure kernel launch
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    MSEKernel<<<blocksPerGrid, threadsPerBlock *sizeof(float)>>>(d_yTrue, d_yPred, d_loss, size);
    hipMemcpy(&h_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_yTrue);
    hipFree(d_yPred);
    hipFree(d_loss);

    // Compute and return the average loss
    return h_loss / size;
}

float gpuCrossEntropyLoss(const float* yTrue, const float* yPred, int batchSize, int numClasses) {
    float* d_yTrue;
    float* d_yPred;
    float* d_loss;
    float h_loss = 0.0f;

    hipMalloc(&d_yTrue, sizeof(float) * batchSize * numClasses);
    hipMalloc(&d_yPred, sizeof(float) * batchSize * numClasses);
    hipMalloc(&d_loss, sizeof(float));

    hipMemcpy(d_yTrue, yTrue, sizeof(float) * batchSize * numClasses, hipMemcpyHostToDevice);
    hipMemcpy(d_yPred, yPred, sizeof(float) * batchSize * numClasses, hipMemcpyHostToDevice);
    hipMemcpy(d_loss, &h_loss, sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (batchSize * numClasses + threadsPerBlock - 1) / threadsPerBlock;

    crossEntropyLossKernel<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(float)>>>(d_yTrue, d_yPred, d_loss, numClasses, batchSize);

    hipMemcpy(&h_loss, d_loss, sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_yTrue);
    hipFree(d_yPred);
    hipFree(d_loss);

    return h_loss / batchSize;
}


}