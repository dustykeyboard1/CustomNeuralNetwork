#include "hip/hip_runtime.h"
#include "MatrixOps.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

namespace MatrixOps {
__global__ void addKernel(const float* A, const float* B, float* C, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void subtractKernel(const float* A, const float* B, float* C, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        C[idx] = A[idx] - B[idx];
    }
}

__global__ void MultiplicationKernel(const float* A, const float* B, float* C, int rowsA, int colsA, int rowsB, int colsB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rowsA && col < colsB) {
        float sum = 0.0f;

        for(int i = 0; i < colsA; ++i) {
            sum += A[row * colsA + i] * B[i * colsB + col];
        }
        C[row * colsB + col] = sum;
    }
}

__global__ void DivideKernel(const float* A, const float* B, float* C, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x; 
    int idx = row * cols + col; 

    if (row < rows && col < cols) {
        if (B[idx] != 0) {  
            C[idx] = A[idx] / B[idx];
        } else {
            C[idx] = 0; 
        }
    }
}

__global__ void TransposeKernel(const float* A, float* B, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < rows && col < cols) {
        int idx = row * cols + col; 
        int tranposed_idx = col * rows + row;
        B[tranposed_idx] = A[idx];
    }

} 

__global__ void ScalerAddKernel(const float* A, float* B, float k, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        B[idx] = A[idx] + k;
    }
}

__global__ void ScalerMultiplyKernel(const float* A, float* B, float k, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        B[idx] = A[idx] * k;
    }
}

__global__ void ReluKernel(const float* A, float* B, int rows, int cols) { 
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        B[idx] = max(A[idx], 0.0f);
    }
}

__global__ void SigmoidKernel(const float* A, float* B, int rows, int cols) { 
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        B[idx] = 1 / (1 + expf(-A[idx]));
    }
}

__global__ void TanhKernel(const float* A, float* B, int rows, int cols) { 
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;

    if (row < rows && col < cols) {
        float exp_pos = expf(A[idx]);  
        float exp_neg = 1.0f / exp_pos; 
        B[idx] = (exp_pos - exp_neg) / (exp_pos + exp_neg);
    }
}

__global__ void SoftMaxKernel(const float* A, float* B, int rows, int cols) { 
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < rows) {
        float rowMax = -FLT_MAX;
        for (int col = 0; col < cols; ++col) {
            rowMax = max(rowMax, A[row * cols + col]);
        }

        float sum = 0.0f;
        for (int col = 0; col < cols; ++col) {
            float expVal = expf(A[row*cols+col]);
            B[row * cols + col] = expVal;
            sum += expVal;

        }
        for (int col = 0; col < cols; ++col) {
            B[row * cols + col] /= sum;
        }
    }
}

__global__ void initializeWeightsKernel(float* weights, int rows, int cols, hiprandState* states, int initTypeCode) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;

    if (row < rows && col < cols) {

        hiprandState localState = states[idx];
        float n_in = rows;  
        float n_out = cols; 

        if (initTypeCode == 1) {
            float limit = sqrtf(6.0f / (n_in + n_out));
            weights[idx] = hiprand_uniform(&localState) * 2.0f * limit - limit;
        } else if (initTypeCode == 2) {
            float stddev = sqrtf(2.0f / n_in);
            weights[idx] = hiprand_normal(&localState) * stddev; 
        } else {
            // Default: Uniform small random values
            weights[idx] = hiprand_uniform(&localState) * 0.01f - 0.005f;
        }
        states[idx] = localState;
    }
}

__global__ void SumAcrossRowsKernel(const float* input, float* output, int rows, int cols) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < cols) {
        float sum = 0.0f;
        for (int row = 0; row < rows; ++row) {
            sum += input[row * cols + col]; // Accumulate values from each row
        }
        output[col] = sum;

    }
}



__global__ void initializeCurandStates(hiprandState* states, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

__global__ void AddBiasKernel(const float* output, const float* bias, float* result, int batchSize, int outputSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < batchSize * outputSize) {
        int col = idx % outputSize; // Column index
        result[idx] = output[idx] + bias[col];
    }
}

void add(const float* A, const float* B, float* C, int rows, int cols) {
    size_t size = rows * cols * sizeof(float);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    addKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rows, cols);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost); 

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

void subtract(const float* A, const float* B, float* C, int rows, int cols) {
    size_t size = rows * cols * sizeof(float);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    subtractKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rows, cols);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost); 

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


}

void multiply(const float* A, const float* B, float* C, int rowsA, int colsA, int rowsB, int colsB) {
    if (colsA != rowsB) {
        std::cerr << "[ERROR] Matrix dimensions do not align for multiplication.\n";
        return;
    }
    if (A == nullptr) {
        std::cerr << "[ERROR] Host pointer A is null.\n";
        return;
    }

    size_t sizeA = rowsA * colsA * sizeof(float);
    std::cout << "[DEBUG] Calculated sizeA: " << sizeA << " bytes\n";

    size_t sizeB = rowsB * colsB * sizeof(float);
    size_t sizeC = rowsA * colsB * sizeof(float);



    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipError_t allocErr = hipMalloc(&d_A, sizeA);
    if (allocErr != hipSuccess) {
        std::cerr << "[ERROR] hipMalloc failed for d_A: " << hipGetErrorString(allocErr) << "\n";
        return;
    }
    std::cout << "[DEBUG] Verifying device-side Matrix A after hipMemcpy (first 10 values): ";
    float* hostAAfterMemcpy = new float[rowsA * colsA];
    hipMemcpy(hostAAfterMemcpy, d_A, sizeA, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    for (int i = 0; i < 10 && i < rowsA * colsA; ++i) {
        std::cout << hostAAfterMemcpy[i] << " ";
    }
    std::cout << "\n";
    delete[] hostAAfterMemcpy;


    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, sizeC, hipMemcpyHostToDevice);


    // Debug: Inspect A
    // float* hostA = new float[rowsA * colsA];
    // hipMemcpy(hostA, d_A, sizeA, hipMemcpyDeviceToHost);
    // std::cout << "[DEBUG] MAtrixOps: Matrix A (first 10 values): ";
    // for (int i = 0; i < 10 && i < rowsA * colsA; ++i) {
    //     std::cout << hostA[i] << " ";
    // }
    // std::cout << "\n";

    // Debug: Inspect B
    float* hostB = new float[rowsB * colsB];
    hipMemcpy(hostB, d_B, sizeB, hipMemcpyDeviceToHost);
    std::cout << "[DEBUG] MAtrixOps: Matrix B (first 10 values): ";
    for (int i = 0; i < 10 && i < rowsB * colsB; ++i) {
        std::cout << hostB[i] << " ";
    }
    std::cout << "\n";

    // Clean up debug arrays
    delete[] hostA;
    delete[] hostB;

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((colsB + 15) / 16, (rowsA + 15) / 16);

    MultiplicationKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rowsA, colsA, rowsB,
                                                                colsB);

    hipDeviceSynchronize();
    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost); 


    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

void divide(const float* A, const float* B, float* C, int rows, int cols) {
    size_t sizeA = cols * rows * sizeof(float);
    size_t sizeB = cols * rows * sizeof(float);
    size_t sizeC = rows * cols * sizeof(float);



    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, sizeC, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    DivideKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rows, cols);

    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost); 

    //Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

void transpose(const float* A, float* B, int rows, int cols) {
    size_t sizeA = rows * cols * sizeof(float);
    size_t sizeB = cols * rows * sizeof(float);

    float *d_A, *d_B;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);

    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    TransposeKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, rows, cols); 

    hipMemcpy(B, d_B, sizeB, hipMemcpyDeviceToHost); 
    hipFree(d_A);
    hipFree(d_B);
}

void scalerAddition(const float* A, float* B, const float k, int rows, int cols) {
    size_t size = rows * cols * sizeof(float);

    float *d_A, *d_B;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);


    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    ScalerAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, k, rows, cols);

    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost); 
    hipFree(d_A);
    hipFree(d_B);

}

void scalerMultiplication(const float* A, float* B, const float k, int rows, int cols) {
    size_t size = rows * cols * sizeof(float);

    float *d_A, *d_B;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);


    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    ScalerMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, k, rows, cols);

    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost); 

    hipFree(d_A);
    hipFree(d_B);

}
void Relu(const float* A, float* B, int rows, int cols) {
    size_t size = rows * cols * sizeof(float);
    float *d_A, *d_B;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    ReluKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, rows, cols);

    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost); 

    //Free memory
    hipFree(d_A);
    hipFree(d_B);

}

void Sigmoid(const float* A, float* B, int rows, int cols) {
    size_t size = rows * cols * sizeof(float);
    float *d_A, *d_B;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    SigmoidKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, rows, cols);

    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost); 

    //Free memory
    hipFree(d_A);
    hipFree(d_B);
}

void Tanh(const float* A, float* B, int rows, int cols) {
    size_t size = rows * cols * sizeof(float);
    float *d_A, *d_B;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    TanhKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, rows, cols);

    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost); 

    hipFree(d_A);
    hipFree(d_B);
} 

void Softmax(const float* A, float* B, int rows, int cols) {
    size_t size = rows * cols * sizeof(float);
    float *d_A, *d_B;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    SoftMaxKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, rows, cols);

    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost); 

    hipFree(d_A);
    hipFree(d_B);
}

void initializeWeights(float* weights, int rows, int cols, const std::string& initType) {
    size_t size = rows * cols * sizeof(float);
    float *d_weights;

    hipMalloc(&d_weights, size);

    hipMemset(d_weights, 0, size);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    int initTypeCode = 0; 
    if (initType == "xavier") {
        initTypeCode = 1;
    } else if (initType == "he") {
        initTypeCode = 2;
    }

    int totalSize = rows * cols;
    hiprandState* d_states;
    hipMalloc(&d_states, totalSize * sizeof(hiprandState));

    int blockSize = 256;
    int gridSize = (totalSize + blockSize - 1) / blockSize;
    initializeCurandStates<<<gridSize, blockSize>>>(d_states, time(0));

    initializeWeightsKernel<<<blocksPerGrid, threadsPerBlock>>>(d_weights, rows, cols, d_states, initTypeCode);
    hipDeviceSynchronize();

    hipMemcpy(weights, d_weights, size, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_weights);
    hipFree(d_states);
}


void MatrixOps::addBias(const float* output, const float* bias, float* result, int batchSize, int outputSize) {
    size_t totalElements = batchSize * outputSize * sizeof(float);
    size_t biasSize = outputSize * sizeof(float);

    float *d_output = nullptr, *d_bias = nullptr, *d_result = nullptr;

    // Allocate GPU memory
    hipMalloc(&d_output, totalElements);
    hipMalloc(&d_bias, biasSize);
    hipMalloc(&d_result, totalElements);

    hipMemcpy(d_output, output, totalElements, hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, biasSize, hipMemcpyHostToDevice);
    hipMemcpy(d_result, result, totalElements, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (batchSize * outputSize + threadsPerBlock - 1) / threadsPerBlock;

    AddBiasKernel<<<blocksPerGrid, threadsPerBlock>>>(d_output, d_bias, d_result, batchSize, outputSize);
    hipDeviceSynchronize();

    hipMemcpy(result, d_result, totalElements, hipMemcpyDeviceToHost);

    hipFree(d_output);
    hipFree(d_bias);
    hipFree(d_result);
}


void MatrixOps::sumAcrossRows(const float* input, float* output, int rows, int cols) {
    float *d_input = nullptr, *d_output = nullptr;
    size_t inputSize = rows * cols * sizeof(float);
    size_t outputSize = cols * sizeof(float);

    hipMalloc(&d_input, inputSize);
    hipMalloc(&d_output, outputSize);

    hipMemcpy(d_input, input, inputSize, hipMemcpyHostToDevice);
    hipMemcpy(d_output, output, outputSize, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (cols + threadsPerBlock - 1) / threadsPerBlock;

    SumAcrossRowsKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, rows, cols);
    hipDeviceSynchronize();

    hipMemcpy(output, d_output, outputSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}


void MatrixOps::reset() {
    hipError_t err = hipDeviceReset();
    if (err != hipSuccess) {
        std::cerr << "CUDA device reset failed: " << hipGetErrorString(err) << "\n";
    }
}


}