#include "hip/hip_runtime.h"
#include "MatrixOps.h"
#include ""
#include <hip/hip_runtime.h>
#include <iostream>

namespace MatrixOps {
__global__ void addKernel(const float* A, const float* B, float* C, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        C[idx] = A[idx] + B[idx];
    }
}

void add(const float* A, const float* B, float* C, int rows, int cols) {
    size_t size = rows * cols * sizeof(float);

    //Allocate Memory on GPU
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    //Copy Data from CPU to GPU
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);

    //Configure block and grid sizes
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    //Launch Kernel
    addKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rows, cols);

    //Copy Result to GPU
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost); 

    //Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


}

}