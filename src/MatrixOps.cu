#include "hip/hip_runtime.h"
#include "MatrixOps.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

namespace MatrixOps {
__global__ void addKernel(const float* A, const float* B, float* C, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void subtractKernel(const float* A, const float* B, float* C, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        C[idx] = A[idx] - B[idx];
    }
}

__global__ void MultiplicationKernel(const float* A, const float* B, float* C, int rowsA, int colsA, int rowsB, int colsB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rowsA && col < colsB) {
        float sum = 0.0f;

        for(int i = 0; i < colsA; ++i) {
            sum += A[row * colsA + i] * B[i * colsB + col];
        }
        C[row * colsB + col] = sum;
    }
}

__global__ void DivideKernel(const float* A, const float* B, float* C, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x; 
    int idx = row * cols + col; 

    if (row < rows && col < cols) {
        if (B[idx] != 0) {  
            C[idx] = A[idx] / B[idx];
        } else {
            C[idx] = 0; 
        }
    }
}

__global__ void TransposeKernel(const float* A, float* B, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < rows && col < cols) {
        int idx = row * cols + col; 
        int tranposed_idx = col * rows + row;
        B[tranposed_idx] = A[idx];
    }

} 

__global__ void ScalerAddKernel(const float* A, float* B, float k, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        B[idx] = A[idx] + k;
    }
}

__global__ void ScalerMultiplyKernel(const float* A, float* B, float k, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        B[idx] = A[idx] * k;
    }
}

__global__ void ReluKernel(const float* A, float* B, int rows, int cols) { 
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        B[idx] = max(A[idx], 0.0f);
    }
}

__global__ void SigmoidKernel(const float* A, float* B, int rows, int cols) { 
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        B[idx] = 1.0f / (1.0f + expf(-A[idx]));
    }
}

__global__ void TanhKernel(const float* A, float* B, int rows, int cols) { 
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        float exp_pos = expf(A[idx]);
        float exp_neg = expf(-A[idx]);  // Changed from 1.0f/exp_pos for numerical stability
        B[idx] = (exp_pos - exp_neg) / (exp_pos + exp_neg);
    }
}

__global__ void SoftMaxKernel(const float* A, float* B, int rows, int cols) { 
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < rows) {
        // Find max for numerical stability
        float max_val = -INFINITY;
        for (int col = 0; col < cols; ++col) {
            max_val = max(max_val, A[row * cols + col]);
        }
        
        // Compute exp and sum
        float sum = 0.0f;
        for (int col = 0; col < cols; ++col) {
            B[row * cols + col] = expf(A[row * cols + col] - max_val);
            sum += B[row * cols + col];
        }
        
        // Normalize
        for (int col = 0; col < cols; ++col) {
            B[row * cols + col] /= sum;
        }
    }
}

__global__ void initializeWeightsKernel(float* weights, int rows, int cols, hiprandState* states, int initTypeCode) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;

    if (row < rows && col < cols) {

        hiprandState localState = states[idx];
        float n_in = rows;  
        float n_out = cols; 

        if (initTypeCode == 1) {
            float limit = sqrtf(6.0f / (n_in + n_out));
            weights[idx] = hiprand_uniform(&localState) * 2.0f * limit - limit;
        } else if (initTypeCode == 2) {
            float stddev = sqrtf(2.0f / n_in);
            weights[idx] = hiprand_normal(&localState) * stddev; 
        } else {
            // Default: Uniform small random values
            weights[idx] = hiprand_uniform(&localState) * 0.01f - 0.005f;
        }
        states[idx] = localState;
    }
}

__global__ void SumAcrossRowsKernel(const float* input, float* output, int rows, int cols) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < cols) {
        float sum = 0.0f;
        for (int row = 0; row < rows; ++row) {
            sum += input[row * cols + col]; // Accumulate values from each row
        }
        output[col] = sum;

    }
}



__global__ void initializeCurandStates(hiprandState* states, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

__global__ void AddBiasKernel(const float* output, const float* bias, float* result, int batchSize, int outputSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < batchSize * outputSize) {
        int col = idx % outputSize; // Column index
        result[idx] = output[idx] + bias[col];
    }
}

__global__ void ReluGradientKernel(const float* output, float* gradient, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        gradient[idx] = output[idx] > 0.0f ? 1.0f : 0.0f;
    }
}

__global__ void SigmoidGradientKernel(const float* output, float* gradient, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        float sig = output[idx];
        gradient[idx] = sig * (1.0f - sig);
    }
}

__global__ void TanhGradientKernel(const float* output, float* gradient, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        float tanh_x = output[idx];
        gradient[idx] = 1.0f - (tanh_x * tanh_x);
    }
}

__global__ void clipValuesKernel(float* input, float min, float max, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < rows && col < cols) {
        int idx = row * cols + col;
        input[idx] = fminf(fmaxf(input[idx], min), max);
    }
}

void MatrixOps::add(const float* A, const float* B, float* C, int rows, int cols, bool isGPU) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    if (isGPU) {
        // Data already on GPU, just launch kernel
        addKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, rows, cols);
    } else {
        // Handle CPU to GPU case
        size_t size = rows * cols * sizeof(float);
        float *d_A, *d_B, *d_C;
        
        hipMalloc(&d_A, size);
        hipMalloc(&d_B, size);
        hipMalloc(&d_C, size);
        
        hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
        
        addKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rows, cols);
        
        hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
        
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }
    hipDeviceSynchronize();
}

void subtract(const float* A, const float* B, float* C, int rows, int cols, bool isGPU) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    if (isGPU) {
        // Data already on GPU, just launch kernel
        subtractKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, rows, cols);
    } else {
        // Handle CPU to GPU case
        size_t size = rows * cols * sizeof(float);
        float *d_A, *d_B, *d_C;
        
        hipMalloc(&d_A, size);
        hipMalloc(&d_B, size);
        hipMalloc(&d_C, size);
        
        hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
        
        subtractKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rows, cols);
        
        hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
        
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }
    hipDeviceSynchronize();
}

void multiply(const float* A, const float* B, float* C, int rowsA, int colsA, int rowsB, int colsB, bool isGPU) {
    if (colsA != rowsB) {
        std::cerr << "[ERROR] Matrix dimensions do not align for multiplication.\n";
        return;
    }

    size_t sizeC = rowsA * colsB * sizeof(float);
    
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((colsB + 15) / 16, (rowsA + 15) / 16);

    if (isGPU) {
        // All pointers are already on GPU, just launch kernel
        MultiplicationKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, rowsA, colsA, rowsB, colsB);
    } else {
        // Handle CPU to GPU case
        float *d_A, *d_B, *d_C;
        size_t sizeA = rowsA * colsA * sizeof(float);
        size_t sizeB = rowsB * colsB * sizeof(float);
        
        hipMalloc(&d_A, sizeA);
        hipMalloc(&d_B, sizeB);
        hipMalloc(&d_C, sizeC);

        hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

        MultiplicationKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rowsA, colsA, rowsB, colsB);

        hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }
    
    hipDeviceSynchronize();
}

void divide(const float* A, const float* B, float* C, int rows, int cols) {
    size_t sizeA = cols * rows * sizeof(float);
    size_t sizeB = cols * rows * sizeof(float);
    size_t sizeC = rows * cols * sizeof(float);



    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, sizeC, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    DivideKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rows, cols);

    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost); 

    //Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

void transpose(const float* A, float* B, int rows, int cols) {
    size_t sizeA = rows * cols * sizeof(float);
    size_t sizeB = cols * rows * sizeof(float);

    float *d_A, *d_B;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);

    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    TransposeKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, rows, cols); 

    hipMemcpy(B, d_B, sizeB, hipMemcpyDeviceToHost); 
    hipFree(d_A);
    hipFree(d_B);
}

void scalerAddition(const float* A, float* B, const float k, int rows, int cols) {
    size_t size = rows * cols * sizeof(float);

    float *d_A, *d_B;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);


    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    ScalerAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, k, rows, cols);

    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost); 
    hipFree(d_A);
    hipFree(d_B);

}

void scalerMultiplication(const float* A, float* B, float k, int rows, int cols, bool isGPU) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    if (isGPU) {
        // Data already on GPU, just launch kernel
        ScalerMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, k, rows, cols);
    } else {
        // Handle CPU to GPU case
        size_t size = rows * cols * sizeof(float);
        float *d_A, *d_B;
        
        hipMalloc(&d_A, size);
        hipMalloc(&d_B, size);
        
        hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
        
        ScalerMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, k, rows, cols);
        
        hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);
        
        hipFree(d_A);
        hipFree(d_B);
    }
    hipDeviceSynchronize();
}
void Relu(const float* A, float* B, int rows, int cols, bool isGPU) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    if (isGPU) {
        // Data already on GPU, just launch kernel
        ReluKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, rows, cols);
    } else {
        // Handle CPU to GPU case
        size_t size = rows * cols * sizeof(float);
        float *d_A, *d_B;
        
        hipMalloc(&d_A, size);
        hipMalloc(&d_B, size);
        
        hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
        
        ReluKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, rows, cols);
        
        hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);
        
        hipFree(d_A);
        hipFree(d_B);
    }
    
    hipDeviceSynchronize();
}

void Sigmoid(const float* A, float* B, int rows, int cols, bool isGPU) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    if (isGPU) {
        // Data already on GPU, just launch kernel
        SigmoidKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, rows, cols);
    } else {
        // Handle CPU to GPU case
        size_t size = rows * cols * sizeof(float);
        float *d_A, *d_B;
        
        hipMalloc(&d_A, size);
        hipMalloc(&d_B, size);
        
        hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
        
        SigmoidKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, rows, cols);
        
        hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);
        
        hipFree(d_A);
        hipFree(d_B);
    }
    
    hipDeviceSynchronize();
}

void Tanh(const float* A, float* B, int rows, int cols, bool isGPU) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    if (isGPU) {
        // Data already on GPU, just launch kernel
        TanhKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, rows, cols);
    } else {
        // Handle CPU to GPU case
        size_t size = rows * cols * sizeof(float);
        float *d_A, *d_B;
        
        hipMalloc(&d_A, size);
        hipMalloc(&d_B, size);
        
        hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
        
        TanhKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, rows, cols);
        
        hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);
        
        hipFree(d_A);
        hipFree(d_B);
    }
    
    hipDeviceSynchronize();
}

void Softmax(const float* A, float* B, int rows, int cols, bool isGPU) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    if (isGPU) {
        // Data already on GPU, just launch kernel
        SoftMaxKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, rows, cols);
    } else {
        // Handle CPU to GPU case
        size_t size = rows * cols * sizeof(float);
        float *d_A, *d_B;
        
        hipMalloc(&d_A, size);
        hipMalloc(&d_B, size);
        
        hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
        
        SoftMaxKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, rows, cols);
        
        hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);
        
        hipFree(d_A);
        hipFree(d_B);
    }
    
    hipDeviceSynchronize();
}

void initializeWeights(float* d_weights, int rows, int cols, const std::string& initType) {
    size_t size = rows * cols * sizeof(float);
    // float *d_weights;

    // hipMalloc(&d_weights, size);

    hipMemset(d_weights, 0, size);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    int initTypeCode = 0; 
    if (initType == "xavier") {
        initTypeCode = 1;
    } else if (initType == "he") {
        initTypeCode = 2;
    }

    int totalSize = rows * cols;
    hiprandState* d_states;
    hipMalloc(&d_states, totalSize * sizeof(hiprandState));

    int blockSize = 256;
    int gridSize = (totalSize + blockSize - 1) / blockSize;
    initializeCurandStates<<<gridSize, blockSize>>>(d_states, time(0));

    initializeWeightsKernel<<<blocksPerGrid, threadsPerBlock>>>(d_weights, rows, cols, d_states, initTypeCode);
    hipDeviceSynchronize();

    // hipMemcpy(weights, d_weights, size, hipMemcpyDeviceToHost);

    // Free memory
    // hipFree(d_weights);
    hipFree(d_states);
}


void MatrixOps::addBias(const float* output, const float* bias, float* result, int batchSize, int outputSize, bool isGPU) {
    size_t totalElements = batchSize * outputSize * sizeof(float);
    size_t biasSize = outputSize * sizeof(float);

    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((batchSize * outputSize + threadsPerBlock.x - 1) / threadsPerBlock.x);

    if (isGPU) {
        // All pointers are already on GPU, just launch kernel
        AddBiasKernel<<<blocksPerGrid, threadsPerBlock>>>(output, bias, result, batchSize, outputSize);
    } else {
        // Handle CPU to GPU case
        float *d_output = nullptr, *d_bias = nullptr, *d_result = nullptr;

        // Allocate GPU memory
        hipMalloc(&d_output, totalElements);
        hipMalloc(&d_bias, biasSize);
        hipMalloc(&d_result, totalElements);

        // Copy data to GPU
        hipMemcpy(d_output, output, totalElements, hipMemcpyHostToDevice);
        hipMemcpy(d_bias, bias, biasSize, hipMemcpyHostToDevice);

        // Launch kernel
        AddBiasKernel<<<blocksPerGrid, threadsPerBlock>>>(d_output, d_bias, d_result, batchSize, outputSize);

        // Copy result back to host
        hipMemcpy(result, d_result, totalElements, hipMemcpyDeviceToHost);

        // Free GPU memory
        hipFree(d_output);
        hipFree(d_bias);
        hipFree(d_result);
    }
    
    hipDeviceSynchronize();
}


void MatrixOps::sumAcrossRows(const float* input, float* output, int rows, int cols, bool isGPU) {
    dim3 threadsPerBlock(256);
    dim3 blocksPerGrid((cols + threadsPerBlock.x - 1) / threadsPerBlock.x);

    if (isGPU) {
        // Data already on GPU, just launch kernel
        SumAcrossRowsKernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, rows, cols);
    } else {
        // Handle CPU to GPU case
        size_t inputSize = rows * cols * sizeof(float);
        size_t outputSize = cols * sizeof(float);
        float *d_input, *d_output;
        
        hipMalloc(&d_input, inputSize);
        hipMalloc(&d_output, outputSize);
        
        hipMemcpy(d_input, input, inputSize, hipMemcpyHostToDevice);
        
        SumAcrossRowsKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, rows, cols);
        
        hipMemcpy(output, d_output, outputSize, hipMemcpyDeviceToHost);
        
        hipFree(d_input);
        hipFree(d_output);
    }
    hipDeviceSynchronize();
}


void MatrixOps::reset() {
    hipError_t err = hipDeviceReset();
    if (err != hipSuccess) {
        std::cerr << "CUDA device reset failed: " << hipGetErrorString(err) << "\n";
    }
}

void MatrixOps::ReluGradient(const float* output, float* gradient, int rows, int cols, bool isGPU) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    if (isGPU) {
        ReluGradientKernel<<<blocksPerGrid, threadsPerBlock>>>(output, gradient, rows, cols);
    } else {
        size_t size = rows * cols * sizeof(float);
        float *d_output, *d_gradient;
        
        hipMalloc(&d_output, size);
        hipMalloc(&d_gradient, size);
        
        hipMemcpy(d_output, output, size, hipMemcpyHostToDevice);
        
        ReluGradientKernel<<<blocksPerGrid, threadsPerBlock>>>(d_output, d_gradient, rows, cols);
        
        hipMemcpy(gradient, d_gradient, size, hipMemcpyDeviceToHost);
        
        hipFree(d_output);
        hipFree(d_gradient);
    }
    hipDeviceSynchronize();
}

void MatrixOps::SigmoidGradient(const float* output, float* gradient, int rows, int cols, bool isGPU) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    if (isGPU) {
        SigmoidGradientKernel<<<blocksPerGrid, threadsPerBlock>>>(output, gradient, rows, cols);
    } else {
        size_t size = rows * cols * sizeof(float);
        float *d_output, *d_gradient;
        
        hipMalloc(&d_output, size);
        hipMalloc(&d_gradient, size);
        
        hipMemcpy(d_output, output, size, hipMemcpyHostToDevice);
        
        SigmoidGradientKernel<<<blocksPerGrid, threadsPerBlock>>>(d_output, d_gradient, rows, cols);
        
        hipMemcpy(gradient, d_gradient, size, hipMemcpyDeviceToHost);
        
        hipFree(d_output);
        hipFree(d_gradient);
    }
    hipDeviceSynchronize();
}

void MatrixOps::TanhGradient(const float* output, float* gradient, int rows, int cols, bool isGPU) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    if (isGPU) {
        TanhGradientKernel<<<blocksPerGrid, threadsPerBlock>>>(output, gradient, rows, cols);
    } else {
        size_t size = rows * cols * sizeof(float);
        float *d_output, *d_gradient;
        
        hipMalloc(&d_output, size);
        hipMalloc(&d_gradient, size);
        
        hipMemcpy(d_output, output, size, hipMemcpyHostToDevice);
        
        TanhGradientKernel<<<blocksPerGrid, threadsPerBlock>>>(d_output, d_gradient, rows, cols);
        
        hipMemcpy(gradient, d_gradient, size, hipMemcpyDeviceToHost);
        
        hipFree(d_output);
        hipFree(d_gradient);
    }
    hipDeviceSynchronize();
}

void MatrixOps::clipValues(float* input, float min, float max, int rows, int cols, bool isGPU) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);
    
    if (isGPU) {
        clipValuesKernel<<<blocksPerGrid, threadsPerBlock>>>(input, min, max, rows, cols);
    } else {
        float* d_input;
        hipMalloc(&d_input, rows * cols * sizeof(float));
        hipMemcpy(d_input, input, rows * cols * sizeof(float), hipMemcpyHostToDevice);
        
        clipValuesKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, min, max, rows, cols);
        
        hipMemcpy(input, d_input, rows * cols * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_input);
    }
    hipDeviceSynchronize();
}

}