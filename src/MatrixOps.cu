#include "hip/hip_runtime.h"
#include "MatrixOps.h"
#include ""
#include <hip/hip_runtime.h>
#include <iostream>

namespace MatrixOps {
__global__ void addKernel(const float* A, const float* B, float* C, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void subtractKernel(const float* A, const float* B, float* C, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        C[idx] = A[idx] - B[idx];
    }
}

__global__ void MultiplicationKernel(const float* A, const float* B, float* C, int rowsA, int colsA, int rowsB, int colsB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rowsA && col < colsB) {
        float sum = 0.0f;

        for(int i = 0; i < colsA; ++i) {
            sum += A[row * colsA + i] * B[i * colsB + col];
        }
        C[row * colsB + col] = sum;
    }
}

__global__ void DivideKernel(const float* A, const float* B, float* C, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x; 
    int idx = row * cols + col; 

    if (row < rows && col < cols) {
        if (B[idx] != 0) {  
            C[idx] = A[idx] / B[idx];
        } else {
            C[idx] = 0; 
        }
    }
}

__global__ void TransposeKernel(const float* A, float* B, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < rows && col < cols) {
        int idx = row * cols + col; 
        int tranposed_idx = col * rows + row;
        B[tranposed_idx] = A[idx];
    }

} 

__global__ void ScalerAddKernel(const float* A, float* B, float k, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        B[idx] = A[idx] + k;
    }
}

__global__ void ScalerMultiplyKernel(const float* A, float* B, float k, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * cols + col;
    if (row < rows && col < cols) {
        B[idx] = A[idx] * k;
    }
}

void add(const float* A, const float* B, float* C, int rows, int cols) {
    size_t size = rows * cols * sizeof(float);

    //Allocate Memory on GPU
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    //Copy Data from CPU to GPU
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);

    //Configure block and grid sizes
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    //Launch Kernel
    addKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rows, cols);

    //Copy Result to GPU
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost); 

    //Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


}

void subtract(const float* A, const float* B, float* C, int rows, int cols) {
    size_t size = rows * cols * sizeof(float);

    //Allocate Memory on GPU
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    //Copy Data from CPU to GPU
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);

    //Configure block and grid sizes
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    //Launch Kernel
    subtractKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rows, cols);

    //Copy Result to GPU
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost); 

    //Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


}

void multiply(const float* A, const float* B, float* C, int rowsA, int colsA, int rowsB, int colsB) {
    size_t sizeA = colsA * rowsA * sizeof(float);
    size_t sizeB = colsB * rowsB * sizeof(float);
    size_t sizeC = rowsA * colsB * sizeof(float);



    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, sizeC, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((colsB + 15) / 16, (rowsA + 15) / 16);

    MultiplicationKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rowsA, colsA, rowsB,
                                                                colsB);

    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost); 

    //Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

void divide(const float* A, const float* B, float* C, int rows, int cols) {
    size_t sizeA = cols * rows * sizeof(float);
    size_t sizeB = cols * rows * sizeof(float);
    size_t sizeC = rows * cols * sizeof(float);



    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, sizeC, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    DivideKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rows, cols);

    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost); 

    //Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

void transpose(const float* A, float* B, int rows, int cols) {
    size_t sizeA = rows * cols * sizeof(float);
    size_t sizeB = cols * rows * sizeof(float);

    float *d_A, *d_B;
    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);

    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    TransposeKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, rows, cols); 

    hipMemcpy(B, d_B, sizeB, hipMemcpyDeviceToHost); 
    hipFree(d_A);
    hipFree(d_B);
}

void scalerAddition(const float* A, float* B, const float k, int rows, int cols) {
    size_t size = rows * cols * sizeof(float);

    //Allocate Memory on GPU
    float *d_A, *d_B;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);


    //Copy Data from CPU to GPU
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    //Configure block and grid sizes
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    //Launch Kernel
    ScalerAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, k, rows, cols);

    //Copy Result to GPU
    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost); 

    //Free memory
    hipFree(d_A);
    hipFree(d_B);

}

void scalerMultiplication(const float* A, float* B, const float k, int rows, int cols) {
    size_t size = rows * cols * sizeof(float);

    //Allocate Memory on GPU
    float *d_A, *d_B;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);


    //Copy Data from CPU to GPU
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    //Configure block and grid sizes
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);

    //Launch Kernel
    ScalerMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, k, rows, cols);

    //Copy Result to GPU
    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost); 

    //Free memory
    hipFree(d_A);
    hipFree(d_B);

}

}