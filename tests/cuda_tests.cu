#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include "MatrixOps.h"

void testAddBias() {
    const int batchSize = 2;
    const int outputSize = 3;

    float hostOutput[6] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f};
    float hostBias[3] = {0.1f, 0.2f, 0.3f};
    float hostResult[6] = {0};
    float expected[6] = {1.1f, 2.2f, 3.3f, 4.1f, 5.2f, 6.3f};

    float *d_output, *d_bias, *d_result;
    hipMalloc(&d_output, sizeof(hostOutput));
    hipMalloc(&d_bias, sizeof(hostBias));
    hipMalloc(&d_result, sizeof(hostResult));

    hipMemcpy(d_output, hostOutput, sizeof(hostOutput), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, hostBias, sizeof(hostBias), hipMemcpyHostToDevice);

    MatrixOps::addBias(d_output, d_bias, d_result, batchSize, outputSize);

    hipMemcpy(hostResult, d_result, sizeof(hostResult), hipMemcpyDeviceToHost);

    for (int i = 0; i < batchSize * outputSize; ++i) {
        assert(fabs(hostResult[i] - expected[i]) < 1e-5);
    }

    std::cout << "addBias passed.\n";

    hipFree(d_output);
    hipFree(d_bias);
    hipFree(d_result);
}

int main() {
    testAddBias();
    return 0;
}
